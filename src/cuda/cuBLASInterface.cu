#include "cuBLASInterface.cuh"
#include <CUDAcommonUtils.cuh>
#include <cmath>

CUBLASContext CUBLASContext::instance;

CUBLASContext::CUBLASContext()
{
  handleCUBLASError(hipblasCreate(&handle));
}

CUBLASContext::~CUBLASContext()
{
  handleCUBLASError(hipblasDestroy(handle));
}

bool CUBLASContext::allocateMatrix(float** device, int rows, int cols)
{
  const int size = rows*cols * sizeof(float);
  return handleCUDAError(hipMalloc((void**)device, size));
}

bool CUBLASContext::freeMatrix(float* device)
{
  return handleCUDAError(hipFree(device));
}

bool CUBLASContext::setMatrix(const float* host, float* device, int rows, int cols)
{
  if(cols == 1)
  {
    return handleCUBLASError(hipblasSetVector(rows, sizeof(*host), host, 1, device, 1));
  }
  else
  {
    return handleCUBLASError(hipblasSetMatrix(rows, cols, sizeof(*host), host, rows, device, rows));
  }
}

bool CUBLASContext::getMatrix(float* host, const float* device, int rows, int cols)
{
  if(cols == 1)
  {
    return handleCUBLASError(hipblasGetVector(rows, sizeof(*device), device, 1, host, 1));
  }
  else
  {
    return handleCUBLASError(hipblasGetMatrix(rows, cols, sizeof(*device), device, rows, host, rows));
  }
}

bool CUBLASContext::multiplyMatrixVector(float* matrix, float* vector, float* result, int rows, int cols)
{
  float alpha = 1.0, beta = 0.0;
  return handleCUBLASError(hipblasSgemv(handle, HIPBLAS_OP_N, rows, cols, &alpha, matrix, rows, vector, 1, &beta, result, 1));
}
