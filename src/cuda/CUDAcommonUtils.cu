#include <CUDAcommonUtils.cuh>
#include <iostream>

bool handleCUDAErrorImpl(hipError_t result, const char* file, int line)
{
  if(result != hipSuccess)
  {
    std::cerr << "CUDA error in file " << file << ", line " << line << ":\n"
        << "\t";
    switch(result)
    {
      case hipErrorMissingConfiguration:
        std::cerr << "hipErrorMissingConfiguration";
        break;
      case hipErrorOutOfMemory:
        std::cerr << "hipErrorOutOfMemory";
        break;
      case hipErrorNotInitialized:
        std::cerr << "hipErrorNotInitialized";
        break;
      case hipErrorLaunchFailure:
        std::cerr << "hipErrorLaunchFailure";
        break;
      case hipErrorPriorLaunchFailure:
        std::cerr << "hipErrorPriorLaunchFailure";
        break;
      case hipErrorLaunchTimeOut:
        std::cerr << "hipErrorLaunchTimeOut";
        break;
      case hipErrorLaunchOutOfResources:
        std::cerr << "hipErrorLaunchOutOfResources";
        break;
      case hipErrorInvalidDeviceFunction:
        std::cerr << "hipErrorInvalidDeviceFunction";
        break;
      case hipErrorInvalidConfiguration:
        std::cerr << "hipErrorInvalidConfiguration";
        break;
      case hipErrorInvalidDevice:
        std::cerr << "hipErrorInvalidDevice";
        break;
      case hipErrorInvalidValue:
        std::cerr << "hipErrorInvalidValue";
        break;
      case hipErrorInvalidPitchValue:
        std::cerr << "hipErrorInvalidPitchValue";
        break;
      case hipErrorInvalidSymbol:
        std::cerr << "hipErrorInvalidSymbol";
        break;
      case hipErrorMapFailed:
        std::cerr << "hipErrorMapFailed";
        break;
      case hipErrorUnmapFailed:
        std::cerr << "hipErrorUnmapFailed";
        break;
      case cudaErrorInvalidHostPointer:
        std::cerr << "cudaErrorInvalidHostPointer";
        break;
      case hipErrorInvalidDevicePointer:
        std::cerr << "hipErrorInvalidDevicePointer";
        break;
      case hipErrorInvalidTexture:
        std::cerr << "hipErrorInvalidTexture";
        break;
      case cudaErrorInvalidTextureBinding:
        std::cerr << "cudaErrorInvalidTextureBinding";
        break;
      case hipErrorInvalidChannelDescriptor:
        std::cerr << "hipErrorInvalidChannelDescriptor";
        break;
      case hipErrorInvalidMemcpyDirection:
        std::cerr << "hipErrorInvalidMemcpyDirection";
        break;
      case cudaErrorAddressOfConstant:
        std::cerr << "cudaErrorAddressOfConstant";
        break;
      case cudaErrorTextureFetchFailed:
        std::cerr << "cudaErrorTextureFetchFailed";
        break;
      case cudaErrorTextureNotBound:
        std::cerr << "cudaErrorTextureNotBound";
        break;
      case cudaErrorSynchronizationError:
        std::cerr << "cudaErrorSynchronizationError";
        break;
      case cudaErrorInvalidFilterSetting:
        std::cerr << "cudaErrorIvalidFilterSetting";
        break;
      case cudaErrorInvalidNormSetting:
        std::cerr << "cudaErrorInvalidNormSetting";
        break;
      case cudaErrorMixedDeviceExecution:
        std::cerr << "cudaErrorMixedDeviceExecution";
        break;
      case hipErrorDeinitialized:
        std::cerr << "hipErrorDeinitialized";
        break;
      case hipErrorUnknown:
        std::cerr << "hipErrorUnknown";
        break;
      case cudaErrorNotYetImplemented:
        std::cerr << "cudaErrorNotYetImplemented";
        break;
      case cudaErrorMemoryValueTooLarge:
        std::cerr << "cudaErrorMemoryValueTooLarge";
        break;
      case hipErrorInvalidHandle:
        std::cerr << "hipErrorInvalidHandle";
        break;
      case hipErrorNotReady:
        std::cerr << "hipErrorNotReady";
        break;
      case cudaErrorStartupFailure:
        std::cerr << "cudaErrorStartupFailure";
        break;
      case cudaErrorApiFailureBase:
        std::cerr << "cudaErrorApiFailureBase";
        break;
      default:
        std::cerr << "unkown error";
    }
    std::cerr << std::endl;
    return false;
  }
  return true;
}

bool handleCUBLASErrorImpl(hipblasStatus_t result, const char* file, int line)
{
  if(result != HIPBLAS_STATUS_SUCCESS)
  {
    std::cerr << "CUBLAS error in file " << file << ", line " << line << ":\n"
        << "\t";
    switch(result)
    {
      case HIPBLAS_STATUS_NOT_INITIALIZED:
        std::cerr << "the CUDA(tm) Runtime initialization failed";
        break;
      case HIPBLAS_STATUS_ALLOC_FAILED:
        std::cerr << "the resources could not be allocated";
        break;
      case HIPBLAS_STATUS_INVALID_VALUE:
        std::cerr << "invalid function arguments";
        break;
      case HIPBLAS_STATUS_MAPPING_ERROR:
        std::cerr << "there was an error accessing the GPU memory";
        break;
      case HIPBLAS_STATUS_EXECUTION_FAILED:
        std::cerr << "the function failed to launch on the GPU";
        break;
      default:
        std::cerr << "unkown error";
    }
    std::cerr << std::endl;
    return false;
  }
  return true;
}
